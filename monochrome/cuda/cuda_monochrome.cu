#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <stdint.h>
#include <time.h>

#include <hip/hip_runtime.h>

#include <SDL2/SDL.h>
#include <SDL2/SDL_image.h>

__global__
void monochrome(uint32_t *pixs, int w, int h)
{
    unsigned int u = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int v = blockIdx.y*blockDim.y + threadIdx.y;
    SDL_Color *p = (SDL_Color*)(pixs + u + v*h);

    float r = p->r / 255.0f;
    float g = p->g / 255.0f;
    float b = p->b / 255.0f;
    float mono = (0.2125 * r) + (0.7154 * g) + (0.0721 * b);
    uint8_t mono_v = (uint8_t)(mono * 255);

    p->r = mono_v;
    p->g = mono_v;
    p->b = mono_v;
}

// function courtesy of showimage.c from SDL2_image lib.
static void draw_background(SDL_Renderer *renderer, int w, int h)
{
    SDL_Color col[2] = {
        { 0x66, 0x66, 0x66, 0xff },
        { 0x99, 0x99, 0x99, 0xff },
    };
    int i, x, y;
    SDL_Rect rect;

    rect.w = 8;
    rect.h = 8;
    for (y = 0; y < h; y += rect.h) {
        for (x = 0; x < w; x += rect.w) {
            i = (((x ^ y) >> 3) & 1);
            SDL_SetRenderDrawColor(renderer, col[i].r, col[i].g, col[i].b, col[i].a);

            rect.x = x;
            rect.y = y;
            SDL_RenderFillRect(renderer, &rect);
        }
    }
}

int main(int argc, char *argv[])
{
    Uint32 flags = 0;
    int w, h, size, done;
    clock_t clk;

    SDL_Window   *window = NULL;
    SDL_Renderer *renderer = NULL;
    SDL_Texture  *img_texture = NULL;
    SDL_Surface  *img_surface = NULL;
    SDL_Event     event;

    const char *img_name = NULL;

    uint32_t *dev_buf = NULL;
    dim3 thrd_per_block(32,32), block_per_grid;

    if(argc != 2)
    {
        fprintf(stderr, "Usage: %s <image_file> \n", argv[0]);
        return -1;
    }

    img_name = argv[1];

    if (SDL_CreateWindowAndRenderer(0, 0, flags, &window, &renderer) < 0)
    {
        fprintf(stderr, "SDL_CreateWindowAndRenderer() failed: %s\n", SDL_GetError());
        return -2;
    }
    SDL_SetWindowPosition(window, 0, 0);

    img_surface = IMG_Load(img_name);
    if(!img_surface)
    {
        fprintf(stderr, "Failed to load %s.\n", img_name);
        return -3;
    }

    w = img_surface->w;
    h = img_surface->h;
    size = w * h * sizeof(uint32_t);

    hipMalloc((void**)&dev_buf, size);
    hipMemcpy(dev_buf, img_surface->pixels, size, hipMemcpyHostToDevice);

    block_per_grid.x = w / thrd_per_block.x;
    block_per_grid.y = h / thrd_per_block.y;

    clk = clock();

    monochrome<<<block_per_grid, thrd_per_block, 0>>>(dev_buf, w, h);
    hipDeviceSynchronize();

    printf("*** Time: %f\n", (float)(clock() - clk) / CLOCKS_PER_SEC);

    hipMemcpy(img_surface->pixels, dev_buf, size, hipMemcpyDeviceToHost);
    hipFree(dev_buf);

    img_texture = SDL_CreateTextureFromSurface(renderer, img_surface);
    if (!img_texture)
    {
        fprintf(stderr, "Couldn't create texture from surface!\n");
        return -1;
    }

    SDL_SetWindowTitle(window, img_name);
    SDL_SetWindowSize(window, w, h);
    SDL_ShowWindow(window);

    done = 0;
    while(!done)
    {
        while(SDL_PollEvent(&event))
        {
            switch (event.type)
            {
                case SDL_KEYUP:
                    switch (event.key.keysym.sym) {
                        case SDLK_ESCAPE:
                            done = 1;
                        default:
                            break;
                    }
                    break;

                case SDL_QUIT:
                    done = 1;
                    break;

                //refresh whatever happened to the window. I'm just too lazy to handle each in detail...
                case SDL_WINDOWEVENT:
                    draw_background(renderer, w, h);
                    SDL_RenderCopy(renderer, img_texture, NULL, NULL);
                    SDL_RenderPresent(renderer);
                    break;

                default:
                    break;
            }
        }

        SDL_Delay(100);
    }

    SDL_FreeSurface(img_surface);
    SDL_DestroyTexture(img_texture);
    SDL_DestroyWindow(window);
    SDL_DestroyRenderer(renderer);
    SDL_Quit();
    return 0;
}

